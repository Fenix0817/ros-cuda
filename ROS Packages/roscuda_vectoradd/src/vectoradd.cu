#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>


__global__ void add(int *a, int *b, int *c)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}

void random_ints(int *i, int size)
{
	for(int k=0; k<size; k++)
	{
		i[k]=rand()%50;
	}
}


//#define N (2048*64)//(2048*2048)
//#define THREADS_PER_BLOCK 512

int *testmain(int num, int threads) 
{
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = num * sizeof(int);
	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a, num);
	b = (int *)malloc(size); random_ints(b, num);
	c = (int *)malloc(size);
	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	// Launch add() kernel on GPU
	add<<<num/threads,threads>>>(d_a, d_b, d_c);
	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return c;
}
